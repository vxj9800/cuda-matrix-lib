#include "imp_includes.hcu"

// cuBLAS API errors
const char *cublasGetErrorString(hipblasStatus_t err)
{
    switch (err) {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
          return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
          return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
          return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
          return "HIPBLAS_STATUS_ARCH_MISMATCH";
        // case CUBLAS_STATUS_MAPPING_err:
        //   return "CUBLAS_STATUS_MAPPING_err";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
          return "HIPBLAS_STATUS_EXECUTION_FAILED";
        // case CUBLAS_STATUS_INTERNAL_err:
        //   return "CUBLAS_STATUS_INTERNAL_err";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
          return "HIPBLAS_STATUS_NOT_SUPPORTED";
        // case CUBLAS_STATUS_LICENSE_err:
        //   return "CUBLAS_STATUS_LICENSE_err";
      }
      return "<unknown>";
}

  // cuSOLVER API errors
const char *cusolverGetErrorString(hipsolverStatus_t err)
{
    switch (err) {
      case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
      case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
      case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
      case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    //   case CUSOLVER_STATUS_MAPPING_err:
    //     return "CUSOLVER_STATUS_MAPPING_err";
      case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    //   case CUSOLVER_STATUS_INTERNAL_err:
    //     return "CUSOLVER_STATUS_INTERNAL_err";
      case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
      case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
      case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
      case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }
    return "<unknown>";
}

  // cuRAND API errors
const char *curandGetErrorString(hiprandStatus_t err)
{
    switch (err) {
      case HIPRAND_STATUS_VERSION_MISMATCH:
        return "HIPRAND_STATUS_VERSION_MISMATCH";
      case HIPRAND_STATUS_NOT_INITIALIZED:
        return "HIPRAND_STATUS_NOT_INITIALIZED";
      case HIPRAND_STATUS_ALLOCATION_FAILED:
        return "HIPRAND_STATUS_ALLOCATION_FAILED";
    //   case CURAND_STATUS_TYPE_err:
    //     return "CURAND_STATUS_TYPE_err";
      case HIPRAND_STATUS_OUT_OF_RANGE:
        return "HIPRAND_STATUS_OUT_OF_RANGE";
      case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
        return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
      case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
        return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
      case HIPRAND_STATUS_LAUNCH_FAILURE:
        return "HIPRAND_STATUS_LAUNCH_FAILURE";
      case HIPRAND_STATUS_PREEXISTING_FAILURE:
        return "HIPRAND_STATUS_PREEXISTING_FAILURE";
      case HIPRAND_STATUS_INITIALIZATION_FAILED:
        return "HIPRAND_STATUS_INITIALIZATION_FAILED";
      case HIPRAND_STATUS_ARCH_MISMATCH:
        return "HIPRAND_STATUS_ARCH_MISMATCH";
    //   case CURAND_STATUS_INTERNAL_err:
    //     return "CURAND_STATUS_INTERNAL_err";
    }
    return "<unknown>";
}

// cuFFT API errors
// static const char *cufftGetErrorString(hipfftResult err)
// {
//     switch (err) {
//       case HIPFFT_INVALID_PLAN:
//         return "HIPFFT_INVALID_PLAN";
//       case HIPFFT_ALLOC_FAILED:
//         return "HIPFFT_ALLOC_FAILED";
//       case HIPFFT_INVALID_TYPE:
//         return "HIPFFT_INVALID_TYPE";
//       case HIPFFT_INVALID_VALUE:
//         return "HIPFFT_INVALID_VALUE";
//       case CUFFT_INTERNAL_err:
//         return "CUFFT_INTERNAL_err";
//       case HIPFFT_EXEC_FAILED:
//         return "HIPFFT_EXEC_FAILED";
//       case HIPFFT_SETUP_FAILED:
//         return "HIPFFT_SETUP_FAILED";
//       case HIPFFT_INVALID_SIZE:
//         return "HIPFFT_INVALID_SIZE";
//       case HIPFFT_UNALIGNED_DATA:
//         return "HIPFFT_UNALIGNED_DATA";
//       case HIPFFT_INCOMPLETE_PARAMETER_LIST:
//         return "HIPFFT_INCOMPLETE_PARAMETER_LIST";
//       case HIPFFT_INVALID_DEVICE:
//         return "HIPFFT_INVALID_DEVICE";
//       case CUFFT_PARSE_err:
//         return "CUFFT_PARSE_err";
//       case HIPFFT_NO_WORKSPACE:
//         return "HIPFFT_NO_WORKSPACE";
//       case HIPFFT_NOT_IMPLEMENTED:
//         return "HIPFFT_NOT_IMPLEMENTED";
//       case CUFFT_LICENSE_err:
//         return "CUFFT_LICENSE_err";
//       case HIPFFT_NOT_SUPPORTED:
//         return "HIPFFT_NOT_SUPPORTED";
//     }
//     return "<unknown>";
// }

  // cuSPARSE API errors
// static const char *hipsparseGetErrorString(hipsparseStatus_t err)
// {
//     switch (err) {
//       case HIPSPARSE_STATUS_NOT_INITIALIZED:
//         return "HIPSPARSE_STATUS_NOT_INITIALIZED";
//       case HIPSPARSE_STATUS_ALLOC_FAILED:
//         return "HIPSPARSE_STATUS_ALLOC_FAILED";
//       case HIPSPARSE_STATUS_INVALID_VALUE:
//         return "HIPSPARSE_STATUS_INVALID_VALUE";
//       case HIPSPARSE_STATUS_ARCH_MISMATCH:
//         return "HIPSPARSE_STATUS_ARCH_MISMATCH";
//       case CUSPARSE_STATUS_MAPPING_err:
//         return "CUSPARSE_STATUS_MAPPING_err";
//       case HIPSPARSE_STATUS_EXECUTION_FAILED:
//         return "HIPSPARSE_STATUS_EXECUTION_FAILED";
//       case CUSPARSE_STATUS_INTERNAL_err:
//         return "CUSPARSE_STATUS_INTERNAL_err";
//       case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
//         return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
//     }
//     return "<unknown>";
// }
