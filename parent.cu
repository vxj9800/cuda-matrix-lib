#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
using namespace std;

#include "./cu_mat_lib/cu_matrix.cu"

int main()
{
    // clock_t begin = clock();

    look_for_errors;
    cu_mat a = randn(5,10);
    cu_mat b = randn(3,2);
    cu_mat c = vertcat(a(2,2,6,8),b);
    a.get(); a(2,3,6,7).get(); b.get(); c.get();
    report_errors;

    // clock_t end = clock();
    // double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    // cout << elapsed_secs;
    
    return (0);
}