#include "imp_includes.hcu"

//#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void HandleError( hipError_t err,const char *file,int line )
{
    confirm(err == hipSuccess,hipGetErrorString( err )<<" in "<<file<<" at "<<line<<".");
}

void HandleError( hipblasStatus_t err,const char *file,int line )
{
    confirm(err == HIPBLAS_STATUS_SUCCESS,cublasGetErrorString( err )<<" in "<<file<<" at "<<line<<".");
}

void HandleError( hipsolverStatus_t err,const char *file,int line )
{
    confirm(err == HIPSOLVER_STATUS_SUCCESS,cusolverGetErrorString( err )<<" in "<<file<<" at "<<line<<".");
}

void HandleError( hiprandStatus_t err,const char *file,int line )
{
    confirm(err == HIPRAND_STATUS_SUCCESS,curandGetErrorString( err )<<" in "<<file<<" at "<<line<<".");
}

// static void HandleError( hipfftResult err,const char *file,int line )
// {
//     confirm(err == HIPFFT_SUCCESS,cufftGetErrorString( err )<<" in "<<file<<" at "<<line<<".");
// }

// static void HandleError( hipsparseStatus_t err,const char *file,int line )
// {
//     confirm(err == HIPSPARSE_STATUS_SUCCESS,hipsparseGetErrorString( err )<<" in "<<file<<" at "<<line<<".");
// }
