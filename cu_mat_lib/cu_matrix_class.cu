class cu_matrix
{
    protected:
    size_t n_rows,n_cols;
    double *p=NULL;
    cu_matrix(){}              // Inaccessible default constructor

    public:
        /***** Constructors *****/
        cu_matrix(const initializer_list<initializer_list<double>> mat);    // Single argument constructor
        cu_matrix(const cu_matrix &to_b_copied);                            // Copy constructor
        cu_matrix(size_t r, size_t c);                                      // Two argument constructor

        /***** Operators *****/
        cu_matrix operator*(const cu_matrix b);                             // Matrix multiplication operator
        cu_matrix operator+(const cu_matrix b);                             // Matrix addition operator

        /***** Member functions *****/
        void get();                                                         // Print data
        size_t rows();                                                      // Get number of rows
        size_t cols();                                                      // Get number of columns

        /***** Supported external functions *****/
        friend cu_matrix randn(size_t r, size_t c);                         // Generate a matrix with normalized random numbers
        friend cu_matrix mld(const cu_matrix a, const cu_matrix b);         // Matrix left divide operator

        /***** Destructor *****/
        ~cu_matrix()                                                        // Destructor to free the memory
        {
            // cout << "Destructor called." << endl;
            HANDLE_ERROR( hipFree(p) );
        }
};



/***********************************************************************************************************************/
cu_matrix::cu_matrix(const cu_matrix &to_b_copied) : n_rows(to_b_copied.n_rows), n_cols(to_b_copied.n_cols)     // Copy constructor
{
    HANDLE_ERROR( hipMalloc((void**)&p,n_rows*n_cols*sizeof(double)) ); // Allocate memory on GPU.
    HANDLE_ERROR( hipMemcpy(p,to_b_copied.p,n_rows*n_cols*sizeof(double),hipMemcpyDeviceToDevice) ); // Copy array from CPU to GPU
}
/***********************************************************************************************************************/


/***********************************************************************************************************************/
cu_matrix::cu_matrix(const initializer_list<initializer_list<double>> mat) : n_rows(mat.size()), n_cols(mat.begin()->size())    // Single argument constructor
// ' -> ' Means:  pointer to an object -> member function. Essentially accessing a member function with the help of a pointer to that object.
{
    // Define number of rows from the array input. Define number of columns from first row of array input
    // Check if the number of elements in each row are same.
    for(int i = 0; i<n_rows; ++i)
    {
        confirm((mat.begin()+i)->size()!=n_cols,"Error: Object initialization failed. Number of elements in each row must be same.");
    }

    // Copy input array to a new matrix while making it column major.
    double *m = new double[n_rows*n_cols]();    // Allocate space on CPU memory.
    confirm(!m,"Error: Memory allocation failed while initializing the object."); // Check proper allocation.                              

    for(int i = 0; i<n_rows; ++i)
    {
        for(int j = 0; j<n_cols; ++j)
        {
            m[j*n_rows+i] = *((mat.begin()+i)->begin()+j);
        }
    }

    HANDLE_ERROR( hipMalloc((void**)&p, n_rows*n_cols*sizeof(double)) ); // Allocate memory on GPU.
    HANDLE_ERROR( hipMemcpy(p,m,n_rows*n_cols*sizeof(double),hipMemcpyHostToDevice) ); // Copy array from CPU to GPU
    delete[] m;
}
/***********************************************************************************************************************/


/***********************************************************************************************************************/
cu_matrix::cu_matrix(size_t r, size_t c) : n_rows(r), n_cols(c)                                      // Two argument constructor
{
    HANDLE_ERROR( hipMalloc((void**)&p, n_rows*n_cols*sizeof(double)) );
}
/***********************************************************************************************************************/


/***********************************************************************************************************************/
cu_matrix cu_matrix::operator*(const cu_matrix b)
{
    confirm(n_cols != b.n_rows,"Error : Matrix multiplication is not possible. Inner matrix dimensions must agree.");
    cu_matrix c(n_rows,b.n_cols);
    double alf = 1.0, bet = 0;
    hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,n_rows,b.n_cols,n_cols,&alf,p,n_rows,b.p,n_cols,&bet,c.p,n_rows);
    hipblasDestroy(handle);
    return c;
}
/***********************************************************************************************************************/


/**********************************************************************************************************************/
// cu_matrix cu_matrix::operator+(const cu_matrix b);                             // Matrix addition operator
// {
//     confirm((n_rows != b.n_rows) && (n_cols != b.n_cols),"Error : Matrix addition is not possible. Matrices must have same dimensions.");

// }
/**********************************************************************************************************************/


/***********************************************************************************************************************/
void cu_matrix::get()   // Print data
{
    double *m = new double[n_rows*n_cols]();    // Allocate space on CPU memory.
    confirm(!m,"Error: Memory allocation failed in 'get()'.") // Check proper allocation.

    // Copy data from GPU to CPU.
    HANDLE_ERROR( hipMemcpy(m,p,n_rows*n_cols*sizeof(double),hipMemcpyDeviceToHost) );
    for(int i = 0; i<n_rows; ++i)
    {
        for(int j = 0; j<n_cols; ++j)
        {
            cout<<" "<<m[j*n_rows+i];
        }
        cout<<endl;
    }
    cout<<endl;
    delete[] m;
}
/***********************************************************************************************************************/


/***********************************************************************************************************************/
size_t cu_matrix::rows(){return n_rows;}                                                      // Get number of rows
/***********************************************************************************************************************/


/***********************************************************************************************************************/
size_t cu_matrix::cols(){return n_cols;}                                                      // Get number of columns
/***********************************************************************************************************************/