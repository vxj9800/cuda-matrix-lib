static void HandleError( hipError_t err,const char *file,int line )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
        throw 1;
    }
}

static void HandleError(hipblasStatus_t err,const char *file,int line )
{
    switch (err)
    {
        case HIPBLAS_STATUS_SUCCESS:
        	return;

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            std::cout << "CUBLAS_STATUS_NOT_INITIALIZEDn in " << file << " at line " << line << "." << endl;
            throw 1;

        case HIPBLAS_STATUS_INVALID_VALUE:
            std::cout << "HIPBLAS_STATUS_INVALID_VALUE" << file << " at line " << line << "." << endl;
            throw 1;

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            std::cout << "HIPBLAS_STATUS_ARCH_MISMATCH" << file << " at line " << line << "." << endl;
            throw 1;

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            std::cout << "HIPBLAS_STATUS_INTERNAL_ERROR" << file << " at line " << line << "." << endl;
            throw 1;
    }
   return;
}

static void HandleError(hipsolverStatus_t err,const char *file,int line )
{
    switch (err)
    {
        case HIPSOLVER_STATUS_SUCCESS:
        	return;

        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            std::cout << "HIPSOLVER_STATUS_NOT_INITIALIZED" << file << " at line " << line << "." << endl;
            throw 1;

        case HIPSOLVER_STATUS_INVALID_VALUE:
            std::cout << "HIPSOLVER_STATUS_INVALID_VALUE" << file << " at line " << line << "." << endl;
            throw 1;

        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            std::cout << "HIPSOLVER_STATUS_ARCH_MISMATCH" << file << " at line " << line << "." << endl;
            throw 1;

        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            std::cout << "HIPSOLVER_STATUS_INTERNAL_ERROR" << file << " at line " << line << "." << endl;
            throw 1;
    }
   return;
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))