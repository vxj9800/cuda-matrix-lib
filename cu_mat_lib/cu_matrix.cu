// Include CUDA libraries
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <hipsolver.h>

// Macro definitions
#define confirm(cond,err)               \
try{                                    \
    if (cond)                           \
    {                                   \
        std::cout << err << endl;       \
        throw 1;                        \
    }                                   \
}                                       \
catch(int n){}

// Include cu_matrix files
#include "cu_error_list.cu"
#include "error_check.cu"
#include "cu_matrix_class.cu"
#include "cu_matrix_functions.cu"