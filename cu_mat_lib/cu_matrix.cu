// Include CUDA libraries
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <hipsolver.h>

// Macro definitions
#define look_for_errors try{

#define report_errors }catch(int n){}

#define confirm(cond,err)                   \
if(!(cond))                                 \
{                                           \
    std::cout << "\a" << err << endl;       \
    throw 1;                                \
}

// Include cu_matrix files
#include "block_dim.cu"
#include "cu_error_list.cu"
#include "error_check.cu"
#include "./class/cu_matrix_class.cu"