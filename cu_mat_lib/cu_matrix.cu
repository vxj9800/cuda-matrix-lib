#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "error_check.cu"
#include "cu_matrix_class.cu"
#include "cu_matrix_functions.cu"