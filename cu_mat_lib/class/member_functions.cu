#include "hip/hip_runtime.h"
#ifndef _CU_MATRIX_CLASS_MEMBER_FUNCTIONS_INCLUDED_
#define _CU_MATRIX_CLASS_MEMBER_FUNCTIONS_INCLUDED_

/************************************   Element wise multiplication   ***********************************************/
__global__ void mat_multiplication(double* a, double* b, double* c, size_t n_ele)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx<n_ele)
    c[idx] = a[idx] * b[idx];
}
cu_mat cu_mat::mult(cu_mat b)
{
    confirm((n_rows == b.n_rows) && (n_cols == b.n_cols),"Error : Matrix multiplication is not possible. Matrices must have same dimensions.");
    cu_mat c(n_rows,n_cols);
    size_t n_ele = n_rows*n_cols;
    size_t n_threads = block_dim(n_ele);
    addition<<<n_ele/n_threads,n_threads>>>(p,b.p,c.p,n_ele);
    HANDLE_ERROR( hipPeekAtLastError() );
    return c;
}
/***********************************************************************************************************************/


/************************************   Print matrix data   ***********************************************/
void cu_mat::get()
{
    double *m = new double[n_rows*n_cols]();    // Allocate space on CPU memory.
    confirm(m,"Error: Memory allocation failed in 'get()'.") // Check proper allocation.

    // Copy data from GPU to CPU.
    HANDLE_ERROR( hipMemcpy(m,p,n_rows*n_cols*sizeof(double),hipMemcpyDeviceToHost) );
    for(int i = 0; i<n_rows; ++i)
    {
        for(int j = 0; j<n_cols; ++j)
        {
            cout<<" "<<m[j*n_rows+i];
        }
        cout<<endl;
    }
    cout<<endl;
    delete[] m;
}
/***********************************************************************************************************************/


/***************************************   Get number of rows   *****************************************/
size_t cu_mat::rows(){return n_rows;}
/***********************************************************************************************************************/


/***************************************   Get number of columns   *****************************************/
size_t cu_mat::cols(){return n_cols;}
/***********************************************************************************************************************/

#endif