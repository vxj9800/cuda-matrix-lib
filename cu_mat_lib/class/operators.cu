#include "hip/hip_runtime.h"
#ifndef _CU_MATRIX_CLASS_OPERATORS_INCLUDED_
#define _CU_MATRIX_CLASS_OPERATORS_INCLUDED_

/**************************************   Access single element of the matrix   *******************************************/
cu_mat cu_mat::operator()(const size_t r, const size_t c)
{
    confirm((r<=n_rows)&&(c<=n_cols),"Error: Index exceeds matrix bounds. The size of the matrix is " << n_rows << "x" << n_cols << ".");
    cu_mat temp(1,1);
    HANDLE_ERROR( hipMemcpy(temp.p,p+(c-1)*n_rows+r-1,sizeof(double),hipMemcpyDeviceToDevice) ); // Copy value from GPU to GPU
    return temp;
}
/***********************************************************************************************************************/


/**************************************   Access sub-matrix   *******************************************/
__global__ void submat(double* dest, double* src, size_t bias, size_t dest_rows, size_t main_rows_bias, size_t n_ele)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx<n_ele)
    dest[idx] = src[bias+idx+idx/dest_rows*main_rows_bias];
}
cu_mat cu_mat::operator()(const size_t r_begin, const size_t r_end, const size_t c_begin, const size_t c_end)
{
    confirm((r_end<=n_rows)&&(c_end<=n_cols),"Error: Index exceeds matrix bounds. The size of the matrix is " << n_rows << "x" << n_cols << ".")
    cu_mat temp(r_end-r_begin+1,c_end-c_begin+1);
    size_t bias = (c_begin-1)*n_rows+r_begin-1;
    size_t main_rows_bias = n_rows-temp.n_rows;
    size_t n_ele = temp.n_rows*temp.n_cols;
    size_t n_threads = block_dim(n_ele);
    submat<<<n_ele/n_threads,n_threads>>>(temp.p,p,bias,temp.n_rows,main_rows_bias,n_ele);
    HANDLE_ERROR( hipPeekAtLastError() );
    return temp;
}
/***********************************************************************************************************************/


/***************************************   Assignment operator to copy 'cu_mat'   **************************************/
cu_mat& cu_mat::operator=(const cu_mat b)
{
    if ((n_rows*n_cols)!=(b.n_rows*b.n_cols))
    {
        HANDLE_ERROR( hipFree(p) );
        HANDLE_ERROR( hipMalloc((void**)&p, b.n_rows*b.n_cols*sizeof(double)) ); // Allocate memory on GPU.
    }
    n_rows = b.n_rows; n_cols = b.n_cols;
    HANDLE_ERROR( hipMemcpy(p,b.p,n_rows*n_cols*sizeof(double),hipMemcpyDeviceToDevice) ); // Copy array from GPU to GPU
    return *this;
}
/***********************************************************************************************************************/


/***************************************   Matrix multiplication   **************************************/
__global__ void const_mat_mult(double *dest, double *src, double *n, size_t n_ele)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx<n_ele)
    dest[idx] = (*n)*src[idx];
}
cu_mat cu_mat::operator*(const cu_mat b)
{
    if ((n_rows)*(n_cols)==1)
    {
        cu_mat c(b.n_rows,b.n_cols);
        size_t n_ele = c.n_rows*c.n_cols, n_threads = block_dim(n_ele);
        const_mat_mult<<<n_ele/n_threads,n_threads>>>(c.p,b.p,p,n_ele);
        return c;
    }
    else if ((b.n_rows)*(b.n_cols)==1)
    {
        cu_mat c(n_rows,n_cols);
        size_t n_ele = c.n_rows*c.n_cols, n_threads = block_dim(n_ele);
        const_mat_mult<<<n_ele/n_threads,n_threads>>>(c.p,p,b.p,n_ele);
        return c;
    }
    else
    {
        confirm(n_cols == b.n_rows,"Error : Matrix multiplication is not possible. Inner matrix dimensions must agree.");
        cu_mat c(n_rows,b.n_cols);
        double alf = 1.0, bet = 0;
        hipblasHandle_t handle;
        HANDLE_ERROR( hipblasCreate(&handle) );
        HANDLE_ERROR( hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,n_rows,b.n_cols,n_cols,&alf,p,n_rows,b.p,n_cols,&bet,c.p,n_rows) );
        HANDLE_ERROR( hipblasDestroy(handle) );
        return c;
    }
}
/***********************************************************************************************************************/


/***************************************   Matrix addition   ****************************************/
__global__ void addition(double* a, double* b, double* c, size_t n_ele)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx<n_ele)
    c[idx] = a[idx] + b[idx];
}
cu_mat cu_mat::operator+(const cu_mat b)                             // Matrix addition operator
{
    confirm((n_rows == b.n_rows) && (n_cols == b.n_cols),"Error : Matrix addition is not possible. Matrices must have same dimensions.");
    cu_mat c(n_rows,n_cols);
    size_t n_ele = n_rows*n_cols;
    size_t n_threads = block_dim(n_ele);
    addition<<<n_ele/n_threads,n_threads>>>(p,b.p,c.p,n_ele);
    HANDLE_ERROR( hipPeekAtLastError() );
    return c;
}
/**********************************************************************************************************************/


/***************************************   Matrix negation   ****************************************/
__global__ void negate_mat(double* a, double* b, double* c, size_t n_ele)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx<n_ele)
    c[idx] = a[idx] - b[idx];
}
cu_mat cu_mat::operator-(const cu_mat b)                             // Matrix negation operator
{
    confirm((n_rows == b.n_rows) && (n_cols == b.n_cols),"Error : Matrix negation is not possible. Matrices must have same dimensions.");
    cu_mat c(n_rows,n_cols);
    size_t n_ele = n_rows*n_cols;
    size_t n_threads = block_dim(n_ele);
    negate_mat<<<n_ele/n_threads,n_threads>>>(p,b.p,c.p,n_ele);
    HANDLE_ERROR( hipPeekAtLastError() );
    return c;
}
/**********************************************************************************************************************/


/***************************************   Matrix power   **************************************/
#include "friend_functions.cu"
cu_mat cu_mat::operator^(const unsigned int n)
{
    confirm(n_rows == n_cols,"Error: Matrix has to be square for matrix power(^) to be executed.")
    // confirm(n>=0,"Error: So far, only natural numbers are supported for powers.")
    if (n == 0)
    {
        return eye(n_rows,n_cols);
    }
    else if (n == 1)
    // if (n==1)
    {
        return *this;
    }
    else
    {
        cu_mat tmp = *this;
        for(int i = 1; i<n; ++i)
        {
            tmp = tmp*(*this);
        }
        return tmp;
    }
}
/***********************************************************************************************************************/


/***************************************   Type conversion from cu_mat to double   **************************************/
cu_mat::operator double()
{
    confirm((n_rows==1) && (n_cols==1), "Error: Type conversion is only possible in the case of 1x1 matrix.");
    double val;
    // Copy data from GPU to CPU.
    HANDLE_ERROR( hipMemcpy(&val,p,sizeof(double),hipMemcpyDeviceToHost) );
    return val;
}

#endif