#ifndef _CU_MATRIX_CLASS_INCLUDED_
#define _CU_MATRIX_CLASS_INCLUDED_

class cu_mat
{
    protected:
    size_t n_rows=0, n_cols=0;
    double *p=NULL;
    cu_mat(){}              // Inaccessible default constructor
    cu_mat(const size_t r, const size_t c, const double n);  // Two argument constructor with initialization

    public:
        /***** Constructors *****/
        cu_mat(const initializer_list<initializer_list<double>> mat);                    // Single argument constructor with 'double' values
        cu_mat(const initializer_list<initializer_list<cu_mat>> mat);                    // Single argument constructor with 'cu_mat' values
        cu_mat(const double n);                                                          // Single value constructor
        cu_mat(const cu_mat &to_b_copied);                                              // Copy constructor

        /***** Operators *****/
        cu_mat operator()(const size_t r, const size_t c);                                           // Matrix element access
        cu_mat operator()(const size_t r_begin, const size_t r_end, const size_t c_begin, const size_t c_end);   // Sub-matrix access
        cu_mat& operator=(const cu_mat b);                                            // Assignment operator to copy 'cu_mat'
        cu_mat& operator=(const double b);                                            // Assignment operator to copy single 'double' value
        cu_mat& operator=(const initializer_list<initializer_list<double>> b);        // Assignment operator to copy 'double' initializer list
        cu_mat& operator=(const initializer_list<initializer_list<cu_mat>> b);        // Assignment operator to copy 'cu_mat' initializer list
        cu_mat operator*(const cu_mat b);                                             // Matrix multiplication operator
        cu_mat operator+(const cu_mat b);                                             // Matrix addition operator
        cu_mat operator-(const cu_mat b);                                             // Matrix negattion operator
        cu_mat operator^(const unsigned int n);                                       // Matrix power operator
        operator double();                                     // Type conversion from cu_mat to double

        /***** Member functions *****/
        cu_mat mult(cu_mat b);                                                              // Element wise multiplication
        void replace(const size_t r, const size_t c, const double n);                       // Replace an element with a 'double' value
        void replace(const size_t r, const size_t c, const cu_mat mat);                     // Replace an element with a 'cu_mat' value
        void replace(const size_t r_begin, const size_t r_end, const size_t c_begin, const size_t c_end, const cu_mat mat);// Replace submatrix with a 'cu_mat' matrix
        void get();                                                                         // Print data
        size_t rows();                                                                      // Get number of rows
        size_t cols();                                                                      // Get number of columns

        /***** Supported external functions *****/
        friend cu_mat randn(const size_t r, const size_t c);                                         // Generate a matrix with normalized random numbers
        friend cu_mat mld(const cu_mat a, const cu_mat b);                                          // Matrix left divide operator
        friend cu_mat eye(const size_t r, const size_t c);                                           // Generate a non-square identity matrix
        friend cu_mat ones(const size_t r, const size_t c);                                          // Matrix with all values 1
        friend cu_mat zeros(const size_t r, const size_t c);                                        // Matrix with all values 0
        friend cu_mat trans(const cu_mat a);                                                        // Transpose of the matrix
        friend cu_mat horzcat(const cu_mat a, const cu_mat b);                                      // Horizontal concatenation of two matrices
        friend cu_mat vertcat(const cu_mat a, const cu_mat b);                                      // Vertical concatenation of two matrices

        /***** Destructor *****/
        ~cu_mat()                                                                        // Destructor to free the memory
        {
            // cout << "Destructor called." << endl;
            HANDLE_ERROR( hipFree(p) );
        }
};

#include "constructors.cu"
#include "operators.cu"
#include "member_functions.cu"
#include "friend_functions.cu"

#endif