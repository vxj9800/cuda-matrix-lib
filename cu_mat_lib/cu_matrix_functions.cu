#include "hip/hip_runtime.h"
cu_matrix randn(size_t r = 1, size_t c = 1)
{
    cu_matrix a(r,c);
    hiprandGenerator_t prng;
	HANDLE_ERROR( hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW) );
	HANDLE_ERROR( hiprandSetPseudoRandomGeneratorSeed(prng,(unsigned long long) clock()) );
	HANDLE_ERROR( hiprandGenerateNormalDouble(prng,a.p,r*c,0.0,1.0) ); //The number of values requested has to be multiple of 2.
    HANDLE_ERROR( hiprandDestroyGenerator(prng) );
    return a;
}

cu_matrix mld(const cu_matrix a, const cu_matrix b) // Adapted from CUSOLVER_Library.pdf QR examples
{
    confirm(a.n_rows != b.n_rows,"Error: 'mld()' operation cannot be performed. Matrix dimensions must agree.")

    cu_matrix A = a, B = b; // Copy current matrix to a new matrix for calculations.
    double *d_tau = NULL;
    double *d_work = NULL, alf = 1.0;
    int *devInfo = NULL, lwork = 0, info_gpu = 0;
    hipsolverHandle_t cusolver_handle = NULL;
    hipblasHandle_t cublas_handle = NULL;

    // step 1: create cusolver/cublas handle
    HANDLE_ERROR( hipsolverDnCreate(&cusolver_handle) );
    HANDLE_ERROR( hipblasCreate(&cublas_handle) );

    // step 2: allocate required extra memory on GPU.
    HANDLE_ERROR( hipMalloc((void**)&d_tau,sizeof(double)*A.n_cols) );
    HANDLE_ERROR( hipMalloc((void**)&devInfo,sizeof(int)) );

    // step 3: query working space of geqrf and ormqr
    HANDLE_ERROR( hipsolverDnDgeqrf_bufferSize(cusolver_handle,A.n_rows,A.n_cols,A.p,A.n_rows,&lwork) );
    HANDLE_ERROR( hipMalloc((void**)&d_work, sizeof(double)*lwork) );

    // step 4: compute QR factorization
    HANDLE_ERROR( hipsolverDnDgeqrf(cusolver_handle,A.n_rows,A.n_cols,A.p,A.n_rows,d_tau,d_work,lwork,devInfo) );
    HANDLE_ERROR( hipDeviceSynchronize() );
    // check if QR is good or not
    HANDLE_ERROR( hipMemcpy(&info_gpu, devInfo, sizeof(int),hipMemcpyDeviceToHost) );
    confirm(info_gpu != 0,"Error: 'mld()' operation cannot be performed. QR decomposition failed.");

    // step 5: compute Q^T*B (CUSOLVER documentation has typos. Follow LAPACK documentation.)
    HANDLE_ERROR( hipsolverDnDormqr(cusolver_handle,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_T,B.n_rows,B.n_cols,A.n_cols,A.p,A.n_rows,d_tau,B.p,B.n_rows,d_work,lwork,devInfo) );
    HANDLE_ERROR( hipDeviceSynchronize() );
    // check if QR is good or not
    HANDLE_ERROR( hipMemcpy(&info_gpu, devInfo, sizeof(int),hipMemcpyDeviceToHost) );
    confirm(info_gpu != 0,"Error: 'mld()' operation cannot be performed. QR decomposition failed.");

    // step 6: compute x = R \ (Q^T*B)
    HANDLE_ERROR( hipblasDtrsm(cublas_handle,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,A.n_cols,B.n_cols,&alf,A.p,A.n_rows,B.p,A.n_cols) );
    HANDLE_ERROR( hipDeviceSynchronize() );

    // Free resources
    HANDLE_ERROR( hipFree(d_tau) );
    HANDLE_ERROR( hipFree(devInfo) );
    HANDLE_ERROR( hipFree(d_work) );

    HANDLE_ERROR( hipblasDestroy(cublas_handle) );
    HANDLE_ERROR( hipsolverDnDestroy(cusolver_handle) );

    return B;
}