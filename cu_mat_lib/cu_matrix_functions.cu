#include "hip/hip_runtime.h"
cu_matrix randn(size_t r = 1, size_t c = 1)
{
    cu_matrix a(r,c);
    hiprandGenerator_t prng;
	HANDLE_ERROR( hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW) );
	HANDLE_ERROR( hiprandSetPseudoRandomGeneratorSeed(prng,(unsigned long long) clock()) );
	HANDLE_ERROR( hiprandGenerateNormalDouble(prng,a.p,r*c,0.0,1.0) ); //The number of values requested has to be multiple of 2.
    HANDLE_ERROR( hiprandDestroyGenerator(prng) );
    return a;
}

cu_matrix mld(const cu_matrix a, const cu_matrix b) // Adapted from CUSOLVER_Library.pdf QR examples
{
    confirm(a.rows != b.rows,"Error: 'mld()' operation cannot be performed. Matrix dimensions must agree.")

    cu_matrix A = a, B = b; // Copy current matrix to a new matrix for calculations.
    double *d_tau = NULL;
    double *d_work = NULL, alf = 1.0;
    int *devInfo = NULL, lwork = 0, info_gpu = 0;
    hipsolverHandle_t cusolver_handle = NULL;
    hipblasHandle_t cublas_handle = NULL;

    // step 1: create cusolver/cublas handle
    HANDLE_ERROR( hipsolverDnCreate(&cusolver_handle) );
    HANDLE_ERROR( hipblasCreate(&cublas_handle) );

    // step 2: allocate required extra memory on GPU.
    HANDLE_ERROR( hipMalloc((void**)&d_tau,sizeof(double)*A.cols) );
    HANDLE_ERROR( hipMalloc((void**)&devInfo,sizeof(int)) );

    // step 3: query working space of geqrf and ormqr
    HANDLE_ERROR( hipsolverDnDgeqrf_bufferSize(cusolver_handle,A.rows,A.cols,A.p,A.rows,&lwork) );
    HANDLE_ERROR( hipMalloc((void**)&d_work, sizeof(double)*lwork) );

    // step 4: compute QR factorization
    HANDLE_ERROR( hipsolverDnDgeqrf(cusolver_handle,A.rows,A.cols,A.p,A.rows,d_tau,d_work,lwork,devInfo) );
    HANDLE_ERROR( hipDeviceSynchronize() );
    // check if QR is good or not
    HANDLE_ERROR( hipMemcpy(&info_gpu, devInfo, sizeof(int),hipMemcpyDeviceToHost) );
    confirm(info_gpu != 0,"Error: 'mld()' operation cannot be performed. QR decomposition failed.");

    // step 5: compute Q^T*B (CUSOLVER documentation has typos. Follow LAPACK documentation.)
    HANDLE_ERROR( hipsolverDnDormqr(cusolver_handle,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_T,B.rows,B.cols,A.cols,A.p,A.rows,d_tau,B.p,B.rows,d_work,lwork,devInfo) );
    HANDLE_ERROR( hipDeviceSynchronize() );
    // check if QR is good or not
    HANDLE_ERROR( hipMemcpy(&info_gpu, devInfo, sizeof(int),hipMemcpyDeviceToHost) );
    confirm(info_gpu != 0,"Error: 'mld()' operation cannot be performed. QR decomposition failed.");

    // step 6: compute x = R \ (Q^T*B)
    HANDLE_ERROR( hipblasDtrsm(cublas_handle,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,A.cols,B.cols,&alf,A.p,A.rows,B.p,A.cols) );
    HANDLE_ERROR( hipDeviceSynchronize() );

    // Free resources
    HANDLE_ERROR( hipFree(d_tau) );
    HANDLE_ERROR( hipFree(devInfo) );
    HANDLE_ERROR( hipFree(d_work) );

    HANDLE_ERROR( hipblasDestroy(cublas_handle) );
    HANDLE_ERROR( hipsolverDnDestroy(cusolver_handle) );

    return B;
}