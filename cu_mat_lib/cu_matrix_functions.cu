#include "hip/hip_runtime.h"
cu_matrix randn(size_t r = 1, size_t c = 1)
{
    cu_matrix a(r,c);
    hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
	hiprandSetPseudoRandomGeneratorSeed(prng,(unsigned long long) clock());
	hiprandGenerateNormalDouble(prng,a.p,r*c,0.0,1.0); //The number of values requested has to be multiple of 2.
    hiprandDestroyGenerator(prng);
    return a;
}