#include "hip/hip_runtime.h"
cu_matrix randn(size_t r = 1, size_t c = 1)
{
    cu_matrix a(r,c);
    hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
	hiprandSetPseudoRandomGeneratorSeed(prng,(unsigned long long) clock());
	hiprandGenerateNormalDouble(prng,a.p,r*c,0.0,1.0); //The number of values requested has to be multiple of 2.
    hiprandDestroyGenerator(prng);
    return a;
}

cu_matrix mld(const cu_matrix a, const cu_matrix b) // Adapted from CUSOLVER_Library.pdf QR examples
{
    if (a.rows != b.rows)
    {
		std::cout << "Error using mld()." << endl;
        std::cout << "Matrix dimensions must agree." << endl;
		throw 1;
    }
    cu_matrix A = a, B = b; // Copy current matrix to a new matrix for calculations.
    double *d_tau = NULL;
    double *d_work = NULL, alf = 1.0;
    int *devInfo = NULL, lwork = 0, info_gpu = 0;
    hipsolverHandle_t cusolver_handle = NULL;
    hipblasHandle_t cublas_handle = NULL;

    // step 1: create cusolver/cublas handle
    HANDLE_ERROR( hipsolverDnCreate(&cusolver_handle) );
    HANDLE_ERROR( hipblasCreate(&cublas_handle) );

    // step 2: allocate required extra memory on GPU.
    HANDLE_ERROR( hipMalloc((void**)&d_tau,sizeof(double)*A.cols) );
    HANDLE_ERROR( hipMalloc((void**)&devInfo,sizeof(int)) );

    // step 3: query working space of geqrf and ormqr
    HANDLE_ERROR( hipsolverDnDgeqrf_bufferSize(cusolver_handle,A.rows,A.cols,A.p,A.rows,&lwork) );
    HANDLE_ERROR( hipMalloc((void**)&d_work, sizeof(double)*lwork) );

    // step 4: compute QR factorization
    HANDLE_ERROR( hipsolverDnDgeqrf(cusolver_handle,A.rows,A.cols,A.p,A.rows,d_tau,d_work,lwork,devInfo) );
    HANDLE_ERROR( hipDeviceSynchronize() );
    // check if QR is good or not
    HANDLE_ERROR( hipMemcpy(&info_gpu, devInfo, sizeof(int),hipMemcpyDeviceToHost) );
    if (info_gpu != 0) { cout << "QR decomposition failed." << endl; throw 1; }

    // step 5: compute Q^T*B (CUSOLVER documentation has typos. Follow LAPACK documentation.)
    HANDLE_ERROR( hipsolverDnDormqr(cusolver_handle,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_T,B.rows,B.cols,A.cols,A.p,A.rows,d_tau,B.p,B.rows,d_work,lwork,devInfo) );
    HANDLE_ERROR( hipDeviceSynchronize() );
    // check if QR is good or not
    HANDLE_ERROR( hipMemcpy(&info_gpu, devInfo, sizeof(int),hipMemcpyDeviceToHost) );
    if (info_gpu != 0) { cout << "QR decomposition failed." << endl; throw 1; }

    // step 6: compute x = R \ (Q^T*B)
    HANDLE_ERROR( hipblasDtrsm(cublas_handle,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,A.cols,B.cols,&alf,A.p,A.rows,B.p,A.cols) );
    HANDLE_ERROR( hipDeviceSynchronize() );

    // Free resources
    HANDLE_ERROR( hipFree(d_tau) );
    HANDLE_ERROR( hipFree(devInfo) );
    HANDLE_ERROR( hipFree(d_work) );

    HANDLE_ERROR( hipblasDestroy(cublas_handle) );
    HANDLE_ERROR( hipsolverDnDestroy(cusolver_handle) );

    return B;
}